#include "hip/hip_runtime.h"
/*
 * Copyright 2016 - 2021 Marcin Matula
 *
 * This file is part of mtrx.
 *
 * mtrx is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * mtrx is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with mtrx.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "cuda_scale_diagonal.hpp"
#include "cuda_diagonal_add.hpp"
#include "cuda_is_ul_triangular.hpp"
#include "cuda_reduce.hpp"
#include "mtrxCore/types.hpp"

extern "C" __global__ void CudaKernel_SF_diagonalAdd(int m, int n, float* matrix, int lda, float value)
{
  cuda_SF_diagonalAdd(m, n, matrix, lda, value);
}

extern "C" __global__ void CudaKernel_SD_diagonalAdd(int m, int n, double* matrix, int lda, double value)
{
  cuda_SD_diagonalAdd(m, n, matrix, lda, value);
}

extern "C" __global__ void CudaKernel_CF_diagonalAdd(int m, int n, hipComplex* matrix, int lda, hipComplex value)
{
  cuda_CF_diagonalAdd(m, n, matrix, lda, value);
}

extern "C" __global__ void CudaKernel_CD_diagonalAdd(int m, int n, hipDoubleComplex* matrix, int lda, hipDoubleComplex value)
{
  cuda_CD_diagonalAdd(m, n, matrix, lda, value);
}

extern "C" __global__ void CudaKernel_SF_scaleDiagonal(int m, int n, float* matrix, int lda, float factor)
{
  cuda_SF_scaleDiagonal(m, n, matrix, lda, factor);
}

extern "C" __global__ void CudaKernel_SD_scaleDiagonal(int m, int n, double* matrix, int lda, double factor)
{
  cuda_SD_scaleDiagonal(m, n, matrix, lda, factor);
}

extern "C" __global__ void CudaKernel_CF_scaleDiagonal(int m, int n, hipComplex* matrix, int lda, hipComplex factor)
{
  cuda_CF_scaleDiagonal(m, n, matrix, lda, factor);
}

extern "C" __global__ void CudaKernel_CD_scaleDiagonal(int m, int n, hipDoubleComplex* matrix, int lda, hipDoubleComplex factor)
{
  cuda_CD_scaleDiagonal(m, n, matrix, lda, factor);
}

extern "C" __global__ void CudaKernel_SF_isUpperTriangular(int m, int n, float* matrix, int lda, float delta, int* reductionResults)
{
  cuda_isUpperTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_SD_isUpperTriangular(int m, int n, double* matrix, int lda, double delta, int* reductionResults)
{
  cuda_isUpperTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_CF_isUpperTriangular(int m, int n, hipComplex* matrix, int lda, hipComplex delta, int* reductionResults)
{
  cuda_isUpperTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_CD_isUpperTriangular(int m, int n, hipDoubleComplex* matrix, int lda, hipDoubleComplex delta, int* reductionResults)
{
  cuda_isUpperTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_SF_isLowerTriangular(int m, int n, float* matrix, int lda, float delta, int* reductionResults)
{
  cuda_isLowerTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_SD_isLowerTriangular(int m, int n, double* matrix, int lda, double delta, int* reductionResults)
{
  cuda_isLowerTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_CF_isLowerTriangular(int m, int n, hipComplex* matrix, int lda, hipComplex delta, int* reductionResults)
{
  cuda_isLowerTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_CD_isLowerTriangular(int m, int n, hipDoubleComplex* matrix, int lda, hipDoubleComplex delta, int* reductionResults)
{
  cuda_isLowerTriangular(m, n, matrix, lda, delta, reductionResults);
}

extern "C" __global__ void CudaKernel_SI_reduceShm(int m, int n, int* array, int lda, int* reductionResults)
{
  cuda_reduce_shm<int>(m, n, array, lda, reductionResults);
}

extern "C" __global__ void CudaKernel_SF_reduceShm(int m, int n, float* array, int lda, float* reductionResults, mtrx::AccumulationMode mode)
{
  cuda_reduce_shm<float>(m, n, array, lda, reductionResults, mode);
}

extern "C" __global__ void CudaKernel_SD_reduceShm(int m, int n, double* array, int lda, double* reductionResults, mtrx::AccumulationMode mode)
{
  cuda_reduce_shm<double>(m, n, array, lda, reductionResults, mode);
}

extern "C" __global__ void CudaKernel_CF_reduceShm(int m, int n, hipComplex* array, int lda, hipComplex* reductionResults, mtrx::AccumulationMode mode)
{
  cuda_reduce_shm<hipComplex>(m, n, array, lda, reductionResults, mode);
}

extern "C" __global__ void CudaKernel_CD_reduceShm(int m, int n, hipDoubleComplex* array, int lda, hipDoubleComplex* reductionResults, mtrx::AccumulationMode mode)
{
  cuda_reduce_shm<hipDoubleComplex>(m, n, array, lda, reductionResults, mode);
}
